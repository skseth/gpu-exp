
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void foo() {}

int main()
{
  foo<<<1,1>>>();

  hipDeviceSynchronize();
  printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));

  return 0;
}
